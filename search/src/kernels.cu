#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: kbmod-usr
 */

#ifndef KERNELS_CU_
#define KERNELS_CU_
#define GPU_LC_FILTER 1
#define MAX_NUM_IMAGES 140

#include "common.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <float.h>
#include "filtering_kernels.cu"

namespace kbmod {


/*
 * Device kernel that convolves the provided image with the psf
 */
__global__ void convolvePSF(int width, int height,
    float *sourceImage, float *resultImage, float *psf,
    int psfRad, int psfDim, float psfSum, float maskFlag)
{
    // Find bounds of convolution area
    const int x = blockIdx.x*CONV_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*CONV_THREAD_DIM+threadIdx.y;
    if (x < 0 || x > width-1 || y < 0 || y > height-1) return;

    // Read kernel
    float sum = 0.0;
    float psfPortion = 0.0;
    float center = sourceImage[y*width+x];
    if (center != NO_DATA) {
        for (int j = -psfRad; j <= psfRad; j++)
        {
            // #pragma unroll
            for (int i = -psfRad; i <= psfRad; i++)
            {
                if ((x + i >= 0) && (x + i < width) &&
                    (y + j >= 0) && (y + j < height))
                {
                    float currentPixel = sourceImage[(y+j)*width+(x+i)];
                    if (currentPixel != NO_DATA)
                    {
                        float currentPSF = psf[(j+psfRad)*psfDim+(i+psfRad)];
                        psfPortion += currentPSF;
                        sum += currentPixel * currentPSF;
                    }
                }
            }
        }

        resultImage[y*width+x] = (sum*psfSum)/psfPortion;
    } else {
        // Leave masked pixel alone (these could be replaced here with zero)
        resultImage[y*width+x] = NO_DATA; // 0.0
    }
}

extern "C" void deviceConvolve(float *sourceImg, float *resultImg,
    int width, int height, float *psfKernel,
    int psfSize, int psfDim, int psfRadius, float psfSum)
{
    // Pointers to device memory //
    float *deviceKernel;
    float *deviceSourceImg;
    float *deviceResultImg;

    long pixelsPerImage = width*height;
    dim3 blocks(width/CONV_THREAD_DIM+1,height/CONV_THREAD_DIM+1);
    dim3 threads(CONV_THREAD_DIM,CONV_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceKernel, sizeof(float)*psfSize));
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*pixelsPerImage));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*pixelsPerImage));

    checkCudaErrors(hipMemcpy(deviceKernel, psfKernel,
        sizeof(float)*psfSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(deviceSourceImg, sourceImg,
        sizeof(float)*pixelsPerImage, hipMemcpyHostToDevice));

    convolvePSF<<<blocks, threads>>> (width, height, deviceSourceImg,
        deviceResultImg, deviceKernel, psfRadius, psfDim, psfSum, NO_DATA);

    checkCudaErrors(hipMemcpy(resultImg, deviceResultImg,
        sizeof(float)*pixelsPerImage, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceKernel));
    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

// Reads a single pixel from an image buffer
__device__ float readPixel(float* img, int x, int y, int width, int height)
{
    return (x<width && y<height) ? img[y*width+x] : NO_DATA;
}

__device__ float maxMasked(float pixel, float previousMax)
{
    return pixel == NO_DATA ? previousMax : max(pixel, previousMax);
}

__device__ float minMasked(float pixel, float previousMin)
{
    return pixel == NO_DATA ? previousMin : min(pixel, previousMin);
}

/*
 * Reduces the resolution of an image to 1/4 using max pooling
 */
__global__ void pool(int sourceWidth, int sourceHeight, float *source,
    int destWidth, int destHeight, float *dest, short mode)
{
    const int x = blockIdx.x*POOL_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*POOL_THREAD_DIM+threadIdx.y;
    if (x>=destWidth || y>=destHeight) return;
    float mp;
    float pixel;
    if (mode == POOL_MAX) {
        mp = -FLT_MAX;
        pixel = readPixel(source, 2*x,   2*y,   sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y,   sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x,   2*y+1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y+1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        if (mp == -FLT_MAX) mp = NO_DATA;
    } else {
        mp = FLT_MAX;
        pixel = readPixel(source, 2*x,   2*y,   sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y,   sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x,   2*y+1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y+1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        if (mp == FLT_MAX) mp = NO_DATA;
    }

    dest[y*destWidth+x] = mp;
}

extern "C" void devicePool(int sourceWidth, int sourceHeight, float *source,
    int destWidth, int destHeight, float *dest, short mode)
{
    // Pointers to device memory //
    float *deviceSourceImg;
    float *deviceResultImg;

    dim3 blocks(destWidth/POOL_THREAD_DIM+1,destHeight/POOL_THREAD_DIM+1);
    dim3 threads(POOL_THREAD_DIM,POOL_THREAD_DIM);

    int srcPixCount = sourceWidth*sourceHeight;
    int destPixCount = destWidth*destHeight;

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*srcPixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*destPixCount));

    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
        sizeof(float)*srcPixCount, hipMemcpyHostToDevice));

    pool<<<blocks, threads>>> (sourceWidth, sourceHeight, deviceSourceImg,
            destWidth, destHeight, deviceResultImg, mode);

    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
        sizeof(float)*destPixCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}



extern "C" void* encodeImage(float *imageVect, unsigned int vectLength, 
                             int numBytes, float minVal, float maxVal, float scale)
{
    void* deviceVect = NULL;

    if (numBytes < 1)
    {
        // Allocate space for the array of floats. 
        checkCudaErrors(hipMalloc((void **)&deviceVect,
                                   sizeof(float)*vectLength));

        // Copy the image vector.
        checkCudaErrors(hipMemcpy(deviceVect, imageVect,
                                   sizeof(float)*vectLength,
                                   hipMemcpyHostToDevice));
    } else if (numBytes == 1)
    {
        // Use a maximum value that is slightly smaller than maxVal
        // but in the same bucket.
        float safe_max = maxVal - scale/10.0;

        // Do the encoding on the host first.
        unsigned int total_size = sizeof(uint8_t) * vectLength;
        uint8_t* encoded = (uint8_t*)malloc(total_size);
        for (unsigned int i = 0; i < vectLength; ++i)
        {
            float value = imageVect[i];
            if (value == NO_DATA)
            {
                encoded[i] = 0;
            } else {
                value = min(value, safe_max);
                value = max(value, minVal);
                value = (value - minVal) / scale;
                encoded[i] = (uint8_t)(value) + 1;
            }
        }
        
        // Allocate the space on device and do a direct copy.
        checkCudaErrors(hipMalloc((void **)&deviceVect, total_size));
        checkCudaErrors(hipMemcpy(deviceVect, encoded, total_size,
                                   hipMemcpyHostToDevice));

        // Free the local space.
        free(encoded);
    } else if (numBytes == 2)
    {
        // Use a maximum value that is slightly smaller than maxVal
        // but in the same bucket.
        float safe_max = maxVal - scale/10.0;

        // Do the encoding on the host first.
        unsigned int total_size = sizeof(uint16_t) * vectLength;
        uint16_t* encoded = (uint16_t*)malloc(total_size);
        for (unsigned int i = 0; i < vectLength; ++i)
        {
            float value = imageVect[i];
            if (value == NO_DATA)
            {
                encoded[i] = 0;
            } else {
                value = min(value, safe_max);
                value = max(value, minVal);
                value = (value - minVal) / scale;
                encoded[i] = (uint16_t)(value) + 1;
            }
        }
        
        // Allocate the space on device and do a direct copy.
        checkCudaErrors(hipMalloc((void **)&deviceVect, total_size));
        checkCudaErrors(hipMemcpy(deviceVect, encoded, total_size,
                                   hipMemcpyHostToDevice));

        // Free the local space.
        free(encoded);
    }

    return deviceVect;
}

__device__ float readEncodedPixel(void* imageVect, int index, int numBytes,
                                  float minVal, float scale)
{
    float value = (numBytes == 1) ? 
            (float)reinterpret_cast<uint8_t*>(imageVect)[index] :
            (float)reinterpret_cast<uint16_t*>(imageVect)[index];
    float result = (value == 0.0) ? NO_DATA : (value - 1.0) * scale + minVal;
    return result;
}

/*
 * Uses pooling to extend min/max regions without reducing the resolution
 * of the image.
 */
__global__ void pool_in_place(int width, int height, float *source, float *dest,
                              int radius, short mode)
{
    const int x = blockIdx.x * POOL_THREAD_DIM + threadIdx.x;
    const int y = blockIdx.y * POOL_THREAD_DIM + threadIdx.y;
    if (x >= width || y >= height)
        return;

    float mp = NO_DATA;
    float pixel;

    // Compute the bounds over which to pool.
    int xs = max(x - radius, 0);
    int xe = min(x + radius, width - 1);
    int ys = max(y - radius, 0);
    int ye = min(y + radius, height - 1);

    if (mode == POOL_MAX) 
    {
        mp = -FLT_MAX;
        for (int xi = xs; xi <= xe; ++xi)
        {
            for (int yi = ys; yi <= ye; ++yi)
            {
                pixel = source[yi * width + xi];
                mp = (pixel == NO_DATA) ? mp : max(pixel, mp);
            }
        }
        if (mp == -FLT_MAX) mp = NO_DATA;
    } else {
        mp = FLT_MAX;
        for (int xi = xs; xi <= xe; ++xi)
        {
            for (int yi = ys; yi <= ye; ++yi)
            {
                pixel = source[yi * width + xi];
                mp = (pixel == NO_DATA) ? mp : min(pixel, mp);
            }
        }
        if (mp == FLT_MAX) mp = NO_DATA;
    }

    dest[y * width + x] = mp;
}

extern "C" void devicePoolInPlace(int width, int height, float *source, float *dest,
                                  int radius, short mode)
{
    // Pointers to device memory //
    float *deviceSourceImg;
    float *deviceResultImg;

    int pixCount = width * height;
    dim3 blocks(width/POOL_THREAD_DIM + 1, height/POOL_THREAD_DIM + 1);
    dim3 threads(POOL_THREAD_DIM, POOL_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*pixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*pixCount));

    // Copy the source image into GPU memory.
    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
                               sizeof(float)*pixCount,
                               hipMemcpyHostToDevice));

    pool_in_place<<<blocks, threads>>> (width, height, deviceSourceImg,
                                        deviceResultImg, radius, mode);

    // Copy the final image from GPU memory to dest.
    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
                               sizeof(float)*pixCount,
                               hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

/*
 * Searches through images (represented as a flat array of floats) looking for most likely
 * trajectories in the given list. Outputs a results image of best trajectories. Returns a
 * fixed number of results per pixel specified by RESULTS_PER_PIXEL
 * filters results using a sigmaG-based filter and a central-moment filter.
 */ 
__global__ void searchFilterImages(int imageCount, int width, int height,
        void *psiVect, void* phiVect, float *imageTimes,
        searchParameters* params, int trajectoryCount,
        trajectory *trajectories, trajectory *results,
        bool useCorr, baryCorrection *baryCorrs)
{
    // Get origin pixel for the trajectories.
    const unsigned short x = blockIdx.x*THREAD_DIM_X+threadIdx.x;
    const unsigned short y = blockIdx.y*THREAD_DIM_Y+threadIdx.y;

    // Data structures used for filtering.
    float lcArray[MAX_NUM_IMAGES];
    float psiArray[MAX_NUM_IMAGES];
    float phiArray[MAX_NUM_IMAGES];
    int idxArray[MAX_NUM_IMAGES];
    int tmpSortIdx;

    // Create an initial set of best results with likelihood -1.0.
    // We also set (x, y) because they are used in the later python
    // functions.
    trajectory best[RESULTS_PER_PIXEL];
    for (int r = 0; r < RESULTS_PER_PIXEL; ++r)
    {
        best[r].x = x;
        best[r].y = y;
        best[r].lh = -1.0;
    }
    
    // Give up on any trajectories starting outside the image
    if (x >= width || y >= height)
    {
        return;
    }

    const unsigned int pixelsPerImage = width * height;

    // For each trajectory we'd like to search
    for (int t=0; t < trajectoryCount; ++t)
    {
        // Create a trajectory for this search.
        trajectory currentT;
        currentT.x = x;
        currentT.y = y;
        currentT.xVel = trajectories[t].xVel;
        currentT.yVel = trajectories[t].yVel;
        currentT.obsCount = 0;

        float psiSum = 0.0;
        float phiSum = 0.0;

        // Loop over each image and sample the appropriate pixel
        for (int i = 0; i < imageCount; ++i)
        {
            lcArray[i] = 0;
            psiArray[i] = 0;
            phiArray[i] = 0;
            idxArray[i] = i;

            // Predict the trajectory's position.
            float cTime = imageTimes[i];
            int currentX = x + int(currentT.xVel*cTime+0.5);
            int currentY = y + int(currentT.yVel*cTime+0.5);

            // If using barycentric correction, apply it
            // This branch is short, and all threads should
            // have same value of baryCorr, so hopefully
            // performance is OK?
            // Must be before out of bounds check
            if (useCorr) {
                baryCorrection bc = baryCorrs[i];
                currentX = int(x + currentT.xVel*cTime + bc.dx + x*bc.dxdx + y*bc.dxdy + 0.5);
                currentY = int(y + currentT.yVel*cTime + bc.dy + x*bc.dydx + y*bc.dydy + 0.5);
            }
                
            // Test if trajectory goes out of image bounds
            // Branching could be avoided here by setting a
            // black image border and clamping coordinates
            if (currentX >= width || currentY >= height
                || currentX < 0 || currentY < 0)
            {
                continue;
            }

            // Get the Psi and Phi pixel values.
            unsigned int pixel_index = (pixelsPerImage*i + currentY*width + currentX);
            float cPsi = (!params->encodeImg) ? reinterpret_cast<float*>(psiVect)[pixel_index] :
                             readEncodedPixel(psiVect, pixel_index, params->psiNumBytes,
                                              params->minPsiVal, params->psiScale);
            if (cPsi == NO_DATA) continue;
            float cPhi = (!params->encodeImg) ? reinterpret_cast<float*>(phiVect)[pixel_index] :
                             readEncodedPixel(phiVect, pixel_index, params->phiNumBytes,
                                              params->minPhiVal, params->phiScale);

            currentT.obsCount++;
            psiSum += cPsi;
            phiSum += cPhi;
            psiArray[i] = cPsi;
            phiArray[i] = cPhi;
            if (cPhi == 0.0)
            {
                lcArray[i] = 0;
            } else {
                lcArray[i] = cPsi/cPhi;
            }
        }
        currentT.lh = psiSum/sqrt(phiSum);
        currentT.flux = psiSum/phiSum;

        // If we do not have enough observations or a good enough LH score,
        // do not bother with any of the following steps.
        if ((currentT.obsCount < minObservations) || 
            (params->doFilter && currentT.lh < params->minLH))
            continue;

        if (doFilter)
        {
            // Sort the the indexes (idxArray) of lcArray in ascending order.
            for (int j = 0; j < imageCount; j++)
            {
                for (int k = j+1; k < imageCount; k++)
                {
                     if (lcArray[idxArray[j]] > lcArray[idxArray[k]])
                     {
                         tmpSortIdx = idxArray[j];
                         idxArray[j] = idxArray[k];
                         idxArray[k] = tmpSortIdx;
                     }
                }
            }

            // Compute index of the three percentile values in lcArray
            // from the given bounds sGL0, 0.5 (median), and sGL1.
            int minKeepIndex = 0;
            int maxKeepIndex = imageCount - 1;
            int imgCountPlus1 = imageCount + 1;
            const int percentiles[3] = {
                int(imgCountPlus1 * params->sGL_L + 0.5) - 1,
                int(imgCountPlus1 * 0.5 + 0.5) - 1,
                int(imgCountPlus1 * params->sGL_H + 0.5) - 1};

            // Compute the lcValues that at +/- 2*sigmaG from the median.
            // This will be used to filter anything outside that range.
            float sigmaG = params->sigmaGCoeff * (lcArray[idxArray[percentiles[2]]]
                               - lcArray[idxArray[percentiles[0]]]);
            float minValue = lcArray[idxArray[percentiles[1]]] - 2 * sigmaG;
            float maxValue = lcArray[idxArray[percentiles[1]]] + 2 * sigmaG;

            // Find the index of the first value in lcArray greater
            // than or equal to minValue.
            for (int i = 0; i <= percentiles[1]; i++)
            {
                int idx = idxArray[i];
                if (lcArray[idx] >= minValue)
                {
                    minKeepIndex = i;
                    break;
                }
            }
            
            // Find the index of the last value in lcArray less
            // than or equal to maxValue.
            for (int i = percentiles[1] + 1; i < imageCount; i++)
            {
                int idx = idxArray[i];
                if (lcArray[idx] <= maxValue)
                {
                    maxKeepIndex = i;
                } else {
                    break;
                }
            }
            
            // Compute the likelihood and flux of the track based on the filtered
            // observations (ones with minValue <= lc <= maxValue).
            float newPsiSum = 0.0;
            float newPhiSum = 0.0;
            for (int i = minKeepIndex; i < maxKeepIndex + 1; i++)
            {
                int idx = idxArray[i];
                newPsiSum += psiArray[idx];
                newPhiSum += phiArray[idx];
            }
            currentT.lh = newPsiSum/sqrt(newPhiSum);
            currentT.flux = newPsiSum/newPhiSum;
        }

        // Insert the new trajectory into the sorted list of results.
        // Only sort the values with valid likelihoods.
        trajectory temp;
        for (int r = 0; r < RESULTS_PER_PIXEL; ++r)
        {
            if (currentT.lh > best[r].lh &&
                currentT.lh > -1.0)
            {
                temp = best[r];
                best[r] = currentT;
                currentT = temp;
            }
        }
    }
    
    // Copy the sorted list of best results for this pixel into
    // the correct location within the global results vector.
    const int base_index = (y * width + x) * RESULTS_PER_PIXEL;
    for (int r = 0; r < RESULTS_PER_PIXEL; ++r)
    {
        results[base_index + r] = best[r];
    }
}

extern "C" void
deviceSearchFilter(int imageCount, int width, int height,
                   float *psiVect, float* phiVect, float *imageTimes,
                   searchParameters* params,
                   int trajCount, trajectory *trajectoriesToSearch,
                   int resultsCount, trajectory *bestTrajects,
                   bool useCorr, baryCorrection *baryCorrs)
{
    // Allocate Device memory
    trajectory *deviceTests;
    float *deviceImgTimes;
    void *devicePsi;
    void *devicePhi;
    trajectory *deviceSearchResults;
    searchParameters *deviceParams;

    checkCudaErrors(hipMalloc((void **)&deviceParams, sizeof(searchParameters)));
    checkCudaErrors(hipMalloc((void **)&deviceTests, sizeof(trajectory)*trajCount));
    checkCudaErrors(hipMalloc((void **)&deviceImgTimes, sizeof(float)*imageCount));
    checkCudaErrors(hipMalloc((void **)&deviceSearchResults,
        sizeof(trajectory)*resultsCount));

    // Copy the Paramter settings.
    checkCudaErrors(hipMemcpy(deviceParams, params,
            sizeof(searchParameters), hipMemcpyHostToDevice));

    // Copy trajectories to search
    checkCudaErrors(hipMemcpy(deviceTests, trajectoriesToSearch,
            sizeof(trajectory)*trajCount, hipMemcpyHostToDevice));

    // Copy image times
    checkCudaErrors(hipMemcpy(deviceImgTimes, imageTimes,
            sizeof(float)*imageCount, hipMemcpyHostToDevice));

    // Copy (and encode) the images.
    unsigned int vectLength = imageCount * width * height;
    devicePsi = encodeImage(psiVect, vectLength, params->encodeImg ? params->psiNumBytes : -1,
                            params->minPsiVal, params->maxPsiVal, params->psiScale);
    devicePhi = encodeImage(phiVect, vectLength, params->encodeImg ? params->phiNumBytes : -1,
                            params->minPhiVal, params->maxPhiVal, params->phiScale);

    // allocate memory for and copy barycentric corrections
    baryCorrection* deviceBaryCorrs;
    if (useCorr) {
        checkCudaErrors(hipMalloc((void **)&deviceBaryCorrs,
            sizeof(baryCorrection)*imageCount));
        checkCudaErrors(hipMemcpy(deviceBaryCorrs, baryCorrs,
            sizeof(baryCorrection)*imageCount, hipMemcpyHostToDevice));
    }

    dim3 blocks(width/THREAD_DIM_X+1,height/THREAD_DIM_Y+1);
    dim3 threads(THREAD_DIM_X,THREAD_DIM_Y);

    // Launch Search
    searchFilterImages<<<blocks, threads>>> (imageCount, width, height,
         devicePsi, devicePhi, deviceImgTimes, deviceParams,
         trajCount, deviceTests, deviceSearchResults,
         useCorr, deviceBaryCorrs);

    // Read back results
    checkCudaErrors(hipMemcpy(bestTrajects, deviceSearchResults,
                sizeof(trajectory)*resultsCount, hipMemcpyDeviceToHost));

    // Free the on GPU memory.
    checkCudaErrors(hipFree(deviceTests));
    checkCudaErrors(hipFree(deviceImgTimes));
    checkCudaErrors(hipFree(deviceSearchResults));
    checkCudaErrors(hipFree(deviceParams));
    checkCudaErrors(hipFree(devicePsi));
    checkCudaErrors(hipFree(devicePhi));
    if (useCorr){
        checkCudaErrors(hipFree(deviceBaryCorrs));
    }
}

} /* namespace kbmod */

#endif /* KERNELS_CU_ */
