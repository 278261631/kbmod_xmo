#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: kbmod-usr
 */

#ifndef KERNELS_CU_
#define KERNELS_CU_
#define GPU_LC_FILTER 1
#define MAX_NUM_IMAGES 140

#include "common.h"
#include <cmath>
#include "cuda_errors.h"
#include <stdexcept>
#include <stdio.h>
#include <float.h>

namespace search {

extern "C" __device__ __host__ void sigmaGFilteredIndicesCU(float *values, int num_values, float sGL0,
                                                            float sGL1, float sigmaGCoeff, float width,
                                                            int *idxArray, int *minKeepIndex,
                                                            int *maxKeepIndex) {
    // Clip the percentiles to [0.01, 99.99] to avoid invalid array accesses.
    if (sGL0 < 0.0001) sGL0 = 0.0001;
    if (sGL1 > 0.9999) sGL1 = 0.9999;

    // Initialize the index array.
    for (int j = 0; j < num_values; j++) {
        idxArray[j] = j;
    }

    // Sort the the indexes (idxArray) of values in ascending order.
    int tmpSortIdx;
    for (int j = 0; j < num_values; j++) {
        for (int k = j + 1; k < num_values; k++) {
            if (values[idxArray[j]] > values[idxArray[k]]) {
                tmpSortIdx = idxArray[j];
                idxArray[j] = idxArray[k];
                idxArray[k] = tmpSortIdx;
            }
        }
    }

    // Compute the index of each of the percent values in values
    // from the given bounds sGL0, 0.5 (median), and sGL1.
    const int pct_L = int(ceil(num_values * sGL0) + 0.001) - 1;
    const int pct_H = int(ceil(num_values * sGL1) + 0.001) - 1;
    const int median_ind = int(ceil(num_values * 0.5) + 0.001) - 1;

    // Compute the values that are +/- (width * sigmaG) from the median.
    float sigmaG = sigmaGCoeff * (values[idxArray[pct_H]] - values[idxArray[pct_L]]);
    float minValue = values[idxArray[median_ind]] - width * sigmaG;
    float maxValue = values[idxArray[median_ind]] + width * sigmaG;

    // Find the index of the first value >= minValue.
    int start = 0;
    while ((start < median_ind) && (values[idxArray[start]] < minValue)) {
        ++start;
    }
    *minKeepIndex = start;

    // Find the index of the last value <= maxValue.
    int end = median_ind + 1;
    while ((end < num_values) && (values[idxArray[end]] <= maxValue)) {
        ++end;
    }
    *maxKeepIndex = end - 1;
}

__device__ float readEncodedPixel(void *imageVect, int index, int numBytes, const scaleParameters &params) {
    float value = (numBytes == 1) ? (float)reinterpret_cast<uint8_t *>(imageVect)[index]
                                  : (float)reinterpret_cast<uint16_t *>(imageVect)[index];
    float result = (value == 0.0) ? NO_DATA : (value - 1.0) * params.scale + params.minVal;
    return result;
}

/*
 * Searches through images (represented as a flat array of floats) looking for most likely
 * trajectories in the given list. Outputs a results image of best trajectories. Returns a
 * fixed number of results per pixel specified by RESULTS_PER_PIXEL
 * filters results using a sigmaG-based filter and a central-moment filter.
 */
__global__ void searchFilterImages(int imageCount, int width, int height, void *psiVect, void *phiVect,
                                   perImageData image_data, searchParameters params, int trajectoryCount,
                                   trajectory *trajectories, trajectory *results) {
    // Get the x and y coordinates within the search space.
    const int x_i = blockIdx.x * THREAD_DIM_X + threadIdx.x;
    const int y_i = blockIdx.y * THREAD_DIM_Y + threadIdx.y;

    // Check that the x and y coordinates are consistent with the search space.
    const int search_width = params.x_start_max - params.x_start_min;
    const int search_height = params.y_start_max - params.y_start_min;
    if ((x_i >= search_width) || (y_i >= search_height)) {
        return;
    }

    // Get origin pixel for the trajectories in pixel space.
    const int x = x_i + params.x_start_min;
    const int y = y_i + params.y_start_min;
    const unsigned int pixelsPerImage = width * height;

    // Data structures used for filtering.
    float lcArray[MAX_NUM_IMAGES];
    float psiArray[MAX_NUM_IMAGES];
    float phiArray[MAX_NUM_IMAGES];
    int idxArray[MAX_NUM_IMAGES];

    // Create an initial set of best results with likelihood -1.0.
    // We also set (x, y) because they are used in the later python
    // functions.
    trajectory best[RESULTS_PER_PIXEL];
    for (int r = 0; r < RESULTS_PER_PIXEL; ++r) {
        best[r].x = x;
        best[r].y = y;
        best[r].lh = -1.0;
    }

    // For each trajectory we'd like to search
    for (int t = 0; t < trajectoryCount; ++t) {
        // Create a trajectory for this search.
        trajectory currentT;
        currentT.x = x;
        currentT.y = y;
        currentT.xVel = trajectories[t].xVel;
        currentT.yVel = trajectories[t].yVel;
        currentT.obsCount = 0;

        float psiSum = 0.0;
        float phiSum = 0.0;

        // Loop over each image and sample the appropriate pixel
        for (int i = 0; i < imageCount; ++i) {
            lcArray[i] = 0;
            psiArray[i] = 0;
            phiArray[i] = 0;
            idxArray[i] = i;
        }

        // Loop over each image and sample the appropriate pixel
        int num_seen = 0;
        for (int i = 0; i < imageCount; ++i) {
            // Predict the trajectory's position.
            float cTime = image_data.imageTimes[i];
            int currentX = x + int(currentT.xVel * cTime + 0.5);
            int currentY = y + int(currentT.yVel * cTime + 0.5);

            // If using barycentric correction, apply it.
            // Must be before out of bounds check
            if (params.useCorr && (image_data.baryCorrs != nullptr)) {
                baryCorrection bc = image_data.baryCorrs[i];
                currentX = int(x + currentT.xVel * cTime + bc.dx + x * bc.dxdx + y * bc.dxdy + 0.5);
                currentY = int(y + currentT.yVel * cTime + bc.dy + x * bc.dydx + y * bc.dydy + 0.5);
            }

            // Test if trajectory goes out of the image, in which case we do not
            // look up a pixel value for this time step (allowing trajectories to
            // overlap the image for only some of the times).
            if (currentX >= width || currentY >= height || currentX < 0 || currentY < 0) {
                continue;
            }

            // Get the Psi and Phi pixel values.
            unsigned int pixel_index = (pixelsPerImage * i + currentY * width + currentX);
            float cPsi = (params.psiNumBytes <= 0 || image_data.psiParams == nullptr)
                                 ? reinterpret_cast<float *>(psiVect)[pixel_index]
                                 : readEncodedPixel(psiVect, pixel_index, params.psiNumBytes,
                                                    image_data.psiParams[i]);
            if (cPsi == NO_DATA) continue;

            float cPhi = (params.phiNumBytes <= 0 || image_data.phiParams == nullptr)
                                 ? reinterpret_cast<float *>(phiVect)[pixel_index]
                                 : readEncodedPixel(phiVect, pixel_index, params.phiNumBytes,
                                                    image_data.phiParams[i]);
            if (cPhi == NO_DATA) continue;

            if (cPsi != NO_DATA && cPhi != NO_DATA) {
                currentT.obsCount++;
                psiSum += cPsi;
                phiSum += cPhi;
                psiArray[num_seen] = cPsi;
                phiArray[num_seen] = cPhi;
                if (cPhi != 0.0) lcArray[num_seen] = cPsi / cPhi;
                num_seen += 1;
            }
        }
        currentT.lh = psiSum / sqrt(phiSum);
        currentT.flux = psiSum / phiSum;

        // If we do not have enough observations or a good enough LH score,
        // do not bother with any of the following steps.
        if ((currentT.obsCount < params.minObservations) ||
            (params.do_sigmag_filter && currentT.lh < params.minLH))
            continue;

        // If we are doing on GPU filtering, run the sigmaG filter
        // and recompute the likelihoods.
        if (params.do_sigmag_filter) {
            int minKeepIndex = 0;
            int maxKeepIndex = num_seen - 1;
            sigmaGFilteredIndicesCU(lcArray, num_seen, params.sGL_L, params.sGL_H, params.sigmaGCoeff, 2.0,
                                    idxArray, &minKeepIndex, &maxKeepIndex);

            // Compute the likelihood and flux of the track based on the filtered
            // observations (ones in [minKeepIndex, maxKeepIndex]).
            float newPsiSum = 0.0;
            float newPhiSum = 0.0;
            for (int i = minKeepIndex; i <= maxKeepIndex; i++) {
                int idx = idxArray[i];
                newPsiSum += psiArray[idx];
                newPhiSum += phiArray[idx];
            }
            currentT.lh = newPsiSum / sqrt(newPhiSum);
            currentT.flux = newPsiSum / newPhiSum;
        }

        // Insert the new trajectory into the sorted list of results.
        // Only sort the values with valid likelihoods.
        trajectory temp;
        for (int r = 0; r < RESULTS_PER_PIXEL; ++r) {
            if (currentT.lh > best[r].lh && currentT.lh > -1.0) {
                temp = best[r];
                best[r] = currentT;
                currentT = temp;
            }
        }
    }

    // Copy the sorted list of best results for this pixel into
    // the correct location within the global results vector.
    // Note the results index is based on the pixel values in search
    // space (not image space).
    const int base_index = (y_i * search_width + x_i) * RESULTS_PER_PIXEL;
    for (int r = 0; r < RESULTS_PER_PIXEL; ++r) {
        results[base_index + r] = best[r];
    }
}

template <typename T>
void *encodeImage(float *imageVect, int numTimes, int numPixels, scaleParameters *params, bool debug) {
    void *deviceVect = NULL;

    long unsigned int total_size = sizeof(T) * numTimes * numPixels;
    if (debug) {
        printf("Encoding image into %lu bytes/pixel for a total of %lu bytes.\n", sizeof(T), total_size);
    }

    // Do the encoding locally first.
    T *encoded = (T *)malloc(total_size);
    for (int t = 0; t < numTimes; ++t) {
        float safe_max = params[t].maxVal - params[t].scale / 100.0;
        for (int p = 0; p < numPixels; ++p) {
            int index = t * numPixels + p;
            float value = imageVect[index];
            if (value == NO_DATA) {
                encoded[index] = 0;
            } else {
                value = min(value, safe_max);
                value = max(value, params[t].minVal);
                value = (value - params[t].minVal) / params[t].scale + 1.0;
                encoded[index] = static_cast<T>(value);
            }
        }
    }

    // Allocate the space on device and do a direct copy.
    checkCudaErrors(hipMalloc((void **)&deviceVect, total_size));
    checkCudaErrors(hipMemcpy(deviceVect, encoded, total_size, hipMemcpyHostToDevice));

    // Free the local space.
    free(encoded);

    return deviceVect;
}

void *encodeImageFloat(float *imageVect, unsigned int vectLength, bool debug) {
    void *deviceVect = NULL;
    long unsigned int total_size = sizeof(float) * vectLength;

    if (debug) {
        printf("Encoding image as float for a total of %lu bytes.\n", total_size);
    }

    checkCudaErrors(hipMalloc((void **)&deviceVect, total_size));
    checkCudaErrors(hipMemcpy(deviceVect, imageVect, total_size, hipMemcpyHostToDevice));
    return deviceVect;
}

extern "C" void deviceSearchFilter(int imageCount, int width, int height, float *psiVect, float *phiVect,
                                   perImageData img_data, searchParameters params, int trajCount,
                                   trajectory *trajectoriesToSearch, int resultsCount,
                                   trajectory *bestTrajects) {
    // Allocate Device memory
    trajectory *deviceTests;
    float *deviceImgTimes;
    void *devicePsi;
    void *devicePhi;
    trajectory *deviceSearchResults;
    baryCorrection *deviceBaryCorrs = nullptr;
    scaleParameters *devicePsiParams = nullptr;
    scaleParameters *devicePhiParams = nullptr;

    // Check the hard coded maximum number of images against the imageCount.
    if (imageCount > MAX_NUM_IMAGES) {
        throw std::runtime_error("Number of images exceeds GPU maximum.");
    }

    if (params.debug) {
        printf("Allocating %lu bytes for testing grid.\n", sizeof(trajectory) * trajCount);
    }
    checkCudaErrors(hipMalloc((void **)&deviceTests, sizeof(trajectory) * trajCount));

    if (params.debug) {
        printf("Allocating %lu bytes for time data.\n", sizeof(float) * imageCount);
    }
    checkCudaErrors(hipMalloc((void **)&deviceImgTimes, sizeof(float) * imageCount));

    if (params.debug) {
        printf("Allocating %lu bytes for testing grid.\n", sizeof(trajectory) * trajCount);
    }
    checkCudaErrors(hipMalloc((void **)&deviceSearchResults, sizeof(trajectory) * resultsCount));

    // Copy trajectories to search
    checkCudaErrors(hipMemcpy(deviceTests, trajectoriesToSearch, sizeof(trajectory) * trajCount,
                               hipMemcpyHostToDevice));

    // Copy image times
    checkCudaErrors(hipMemcpy(deviceImgTimes, img_data.imageTimes, sizeof(float) * imageCount,
                               hipMemcpyHostToDevice));

    // Copy (and encode) the images. Also copy over the scaling parameters if needed.
    if ((params.psiNumBytes == 1 || params.psiNumBytes == 2) && (img_data.psiParams != nullptr)) {
        checkCudaErrors(hipMalloc((void **)&devicePsiParams, imageCount * sizeof(scaleParameters)));
        checkCudaErrors(hipMemcpy(devicePsiParams, img_data.psiParams, imageCount * sizeof(scaleParameters),
                                   hipMemcpyHostToDevice));
        if (params.psiNumBytes == 1) {
            devicePsi = encodeImage<uint8_t>(psiVect, imageCount, width * height, img_data.psiParams,
                                             params.debug);
        } else {
            devicePsi = encodeImage<uint16_t>(psiVect, imageCount, width * height, img_data.psiParams,
                                              params.debug);
        }
    } else {
        devicePsi = encodeImageFloat(psiVect, imageCount * width * height, params.debug);
    }
    if ((params.phiNumBytes == 1 || params.phiNumBytes == 2) && (img_data.phiParams != nullptr)) {
        checkCudaErrors(hipMalloc((void **)&devicePhiParams, imageCount * sizeof(scaleParameters)));
        checkCudaErrors(hipMemcpy(devicePhiParams, img_data.phiParams, imageCount * sizeof(scaleParameters),
                                   hipMemcpyHostToDevice));
        if (params.phiNumBytes == 1) {
            devicePhi = encodeImage<uint8_t>(phiVect, imageCount, width * height, img_data.phiParams,
                                             params.debug);
        } else {
            devicePhi = encodeImage<uint16_t>(phiVect, imageCount, width * height, img_data.phiParams,
                                              params.debug);
        }
    } else {
        devicePhi = encodeImageFloat(phiVect, imageCount * width * height, params.debug);
    }

    // allocate memory for and copy barycentric corrections
    if (params.useCorr) {
        if (params.debug) {
            printf("Search is using barycentric corrections (%lu bytes).\n",
                   sizeof(baryCorrection) * imageCount);
        }
        checkCudaErrors(hipMalloc((void **)&deviceBaryCorrs, sizeof(baryCorrection) * imageCount));
        checkCudaErrors(hipMemcpy(deviceBaryCorrs, img_data.baryCorrs, sizeof(baryCorrection) * imageCount,
                                   hipMemcpyHostToDevice));
    }

    // Wrap the per-image data into a struct. This struct will be copied by value
    // during the function call, so we don't need to allocate memory for the
    // struct itself. We just set the pointers to the on device vectors.
    perImageData device_image_data;
    device_image_data.numImages = imageCount;
    device_image_data.imageTimes = deviceImgTimes;
    device_image_data.baryCorrs = deviceBaryCorrs;
    device_image_data.psiParams = devicePsiParams;
    device_image_data.phiParams = devicePhiParams;

    // Compute the range of starting pixels to use when setting the blocks and threads.
    // We use the width and height of the search space (as opposed to the image width
    // and height), meaning the blocks/threads will be indexed relative to the search space.
    int search_width = params.x_start_max - params.x_start_min;
    int search_height = params.y_start_max - params.y_start_min;
    dim3 blocks(search_width / THREAD_DIM_X + 1, search_height / THREAD_DIM_Y + 1);
    dim3 threads(THREAD_DIM_X, THREAD_DIM_Y);

    // Launch Search
    searchFilterImages<<<blocks, threads>>>(imageCount, width, height, devicePsi, devicePhi,
                                            device_image_data, params, trajCount, deviceTests,
                                            deviceSearchResults);

    // Read back results
    checkCudaErrors(hipMemcpy(bestTrajects, deviceSearchResults, sizeof(trajectory) * resultsCount,
                               hipMemcpyDeviceToHost));

    // Free the on GPU memory.
    if (deviceBaryCorrs != nullptr) checkCudaErrors(hipFree(deviceBaryCorrs));
    if (devicePhiParams != nullptr) checkCudaErrors(hipFree(devicePhiParams));
    if (devicePsiParams != nullptr) checkCudaErrors(hipFree(devicePsiParams));
    checkCudaErrors(hipFree(devicePhi));
    checkCudaErrors(hipFree(devicePsi));
    checkCudaErrors(hipFree(deviceSearchResults));
    checkCudaErrors(hipFree(deviceImgTimes));
    checkCudaErrors(hipFree(deviceTests));
}

} /* namespace search */

#endif /* KERNELS_CU_ */
